#include "hip/hip_runtime.h"
/*
 *  _reg_mutualinformation_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_MUTUALINFORMATION_GPU_CU
#define _REG_MUTUALINFORMATION_GPU_CU

#include "_reg_blocksize_gpu.h"
#include "_reg_mutualinformation_gpu.h"
#include "_reg_mutualinformation.h"
#include "_reg_tools.h"
#include "_reg_mutualinformation_kernels.cu"
#include <iostream>
#include <sys/time.h>


double GetBasisSplineValue(double x)
{
    x=fabs(x);
    double value=0.0;
    if(x<2.0){
        if(x<1.0)
            value = (double)(2.0f/3.0f + (0.5f*x-1.0)*x*x);
        else{
            x-=2.0f;
            value = -x*x*x/6.0f;
        }
    }
    return value;
}
//template<class double>
void sum_axes(int axes, int current, double *histogram, double *&sums,
              int num_dims, int *dimensions, int *indices)
{
    int index;
    double value = (double)0;

    for(indices[current] = 0; indices[current] < dimensions[current]; ++indices[current])
    {
        if(axes == current) {
            index = calculate_index(num_dims, dimensions, indices);
            value += histogram[index];
        }
        else {
            sum_axes(axes, previous(current, num_dims), histogram,
                                    sums, num_dims, dimensions, indices);
        }
    }
    // Store the sum along the current line and increment the storage pointer
    if (axes == current)
    {
        *(sums) = value;
        ++sums;
    }
}
void smooth_axes(int axes, int current, double *histogram,
                 double *result, double *window,
                 int num_dims, int *dimensions, int *indices)
{
    int temp, index;
    double value;
    for(indices[current] = 0; indices[current] < dimensions[current]; ++indices[current])
    {
        if(axes == current) {
            temp = indices[current];
            indices[current]--;
            value = (double)0;
            for(int it=0; it<3; it++) {
                if(-1<indices[current] && indices[current]<dimensions[current]) {
                    index = calculate_index(num_dims, dimensions, indices);
                    value += histogram[index] * window[it];
                }
                indices[current]++;
            }
            indices[current] = temp;
            index = calculate_index(num_dims, dimensions, indices);
            result[index] = value;
        }
        else {
            smooth_axes(axes, previous(current, num_dims), histogram,
                                       result, window, num_dims, dimensions, indices);
        }
    }
}

/// Traverse the histogram along the specified axes and smooth along it
//template<class double>
void traverse_and_smooth_axes(int axes, double *histogram,
                              double *result, double *window,
                              int num_dims, int *dimensions)
{
    int indices[num_dims];
    for(int dim = 0; dim < num_dims; ++dim) indices[dim] = 0;

    smooth_axes(axes, previous(axes, num_dims), histogram,
                               result, window, num_dims, dimensions, indices);
}

/// Sum along the specified axes. Uses recursion


/// Traverse and sum along an axes
//template<class double>
void traverse_and_sum_axes(int axes, double *histogram, double *&sums,
                           int num_dims, int *dimensions)
{
    int indices[num_dims];
    for(int dim = 0; dim < num_dims; ++dim) indices[dim] = 0;
    sum_axes(axes, previous(axes, num_dims), histogram, sums,
                            num_dims, dimensions, indices);
}



//new gpu getentropy function
void reg_getEntropies_gpu(nifti_image *targetImage,
                      nifti_image *resultImage,
                      unsigned int *target_bins, // should be an array of size num_target_volumes
                      unsigned int *result_bins, // should be an array of size num_result_volumes
                      double *probaJointHistogram,
                      double *logJointHistogram,
                      double *entropies,
                      int *mask,
                      bool approx,
					  float *c_targetImage,
					  float *c_resultImage,
					  int *c_mask,
					  int activeVoxelNumber)
{	

  
	//double *c_targetImage,*c_resultImage;
	//float *c_targetImage = static_cast<float *>(targetImage->data);
	//float *c_resultImage = static_cast<float *>(resultImage->data);
	
	struct timeval t1, t2,t3,t4,t5,t6;
    double elapsedTime,elapsedTime1,elapsedTime2;
	
	int *c_probaJointHistogram_int,*c_probaJointHistogram,*c_voxel_number,*voxel_number_blk;
	int num_target_volumes = targetImage->nt;
    int num_result_volumes = resultImage->nt;
	int i, j;
	int voxel_number=0;
	if(num_target_volumes>1 || num_result_volumes>1) approx=true;
	
    int targetVoxelNumber = targetImage->nx * targetImage->ny * targetImage->nz;
	//int resultVoxelNumber = resultImage->nx * resultImage->ny * resultImage->nz;
	 //fprintf(stderr,"[NiftyReg Debug parag] targetVoxelNumber= %d\n",resultVoxelNumber); 
/*     DTYPE *targetImagePtr = static_cast<DTYPE *>(targetImage->data);
    DTYPE *resultImagePtr = static_cast<DTYPE *>(resultImage->data); */

    // Build up this arrays of offsets that will help us index the histogram entries
/*     SafeArray<int> target_offsets(num_target_volumes);
    SafeArray<int> result_offsets(num_result_volumes); */
	int target_offsets[num_target_volumes];
	int result_offsets[num_result_volumes];
	
	

    int num_histogram_entries = 1;
    int total_target_entries = 1;
    int total_result_entries = 1;
	int nthreads=1024;
	int reduce_size=ceil(targetVoxelNumber/nthreads);
    // Data pointers
    int histogram_dimensions[num_target_volumes + num_result_volumes];

    // Calculate some constants and initialize the data pointers
    for (i = 0; i < num_target_volumes; ++i) {
        num_histogram_entries *= target_bins[i];
        total_target_entries *= target_bins[i];
        histogram_dimensions[i] = target_bins[i];
		/* fprintf(stderr,"[NiftyReg Debug parag] target_bins= %d\n",num_histogram_entries); */
		
        target_offsets[i] = 1;
        for (j = i; j > 0; --j) {
			//fprintf(stderr,"[NiftyReg Debug parag] j= %d\n",j);
			target_offsets[i] *= target_bins[j - 1];
			}
    }

    for (i = 0; i < num_result_volumes; ++i) {
        num_histogram_entries *= result_bins[i];
        total_result_entries *= result_bins[i];
        histogram_dimensions[num_target_volumes + i] = result_bins[i];

        result_offsets[i] = 1;
        for (j = i; j > 0; --j) result_offsets[i] *= result_bins[j-1];
    }
    int num_probabilities = num_histogram_entries;
	num_histogram_entries += total_target_entries + total_result_entries;

	c_probaJointHistogram_int = (int *)malloc(num_histogram_entries * sizeof(int));
	voxel_number_blk=(int *)malloc(reduce_size * sizeof(int));
    memset(c_probaJointHistogram_int, 0, num_histogram_entries * sizeof(int));
	memset(voxel_number_blk, 0, reduce_size * sizeof(int));
    memset(probaJointHistogram, 0, num_histogram_entries * sizeof(double));
	memset(logJointHistogram, 0, num_histogram_entries * sizeof(double));

/* 	NR_CUDA_SAFE_CALL((hipMemcpy(mask, c_mask, targetVoxelNumber * sizeof(int), hipMemcpyDeviceToHost)));
    // Space for storing the marginal entropies.
				for (i=0;i<targetVoxelNumber;i++)
	{
		printf("[NiftyReg Debug parag] index=%d mask=%d\n",i,mask[i]);
		
	} */
	
	 
	 // allocate and initialize an array of stream handles
 
	//allocate memory 
 	
	 gettimeofday(&t1, NULL);
	NR_CUDA_SAFE_CALL(hipMalloc(&c_probaJointHistogram,num_histogram_entries * sizeof(int)));
	//NR_CUDA_SAFE_CALL(hipMalloc(&c_mask,targetVoxelNumber * sizeof(int)));
	//NR_CUDA_SAFE_CALL(hipMalloc(&c_resultImage,resultVoxelNumber * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMalloc(&c_voxel_number,reduce_size * sizeof(int)));
	

/* 	NR_CUDA_SAFE_CALL(hipHostMalloc(&c_probaJointHistogram,num_histogram_entries * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipHostMalloc(&c_targetImage,targetVoxelNumber * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipHostMalloc(&c_resultImage,resultVoxelNumber * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipHostMalloc(&c_voxel_number,reduce_size * sizeof(int))); */
	
	gettimeofday(&t5, NULL);
	NR_CUDA_SAFE_CALL((hipMemcpy(c_probaJointHistogram, c_probaJointHistogram_int, num_histogram_entries * sizeof(int), hipMemcpyHostToDevice)));
	//NR_CUDA_SAFE_CALL((hipMemcpy(c_mask, mask, targetVoxelNumber * sizeof(int), hipMemcpyHostToDevice)));
	//NR_CUDA_SAFE_CALL((hipMemcpy(c_resultImage, resultImage->data, resultVoxelNumber * sizeof(float), hipMemcpyHostToDevice)));
	NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, c_mask, targetVoxelNumber*sizeof(int)));
	//NR_CUDA_SAFE_CALL((hipMemcpy(c_voxel_number, voxel_number_blk, reduce_size * sizeof(int), hipMemcpyHostToDevice)));
	gettimeofday(&t6, NULL);
	
	//NR_CUDA_SAFE_CALL((hipMemcpy(c_targetVoxelNumber, &targetVoxelNumber, sizeof(int), hipMemcpyHostToDevice)));
	dim3 G1(ceil(activeVoxelNumber/nthreads),1,1);
	dim3 B1(nthreads,1,1);
	
	int shared_size = nthreads* sizeof(int);
	shared_size += num_histogram_entries * sizeof(int);
	gettimeofday(&t3, NULL);
	//reg_getJointHistogram_kernel1<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,c_voxel_number,targetVoxelNumber,total_target_entries);
	//reg_getJointHistogram_kernel2<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,c_voxel_number,targetVoxelNumber,total_target_entries);
	//reg_getJointHistogram_kernel3<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,c_voxel_number,targetVoxelNumber,total_target_entries);
	//reg_getJointHistogram_kernel4<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,c_voxel_number,targetVoxelNumber,total_target_entries);
	reg_getJointHistogram_kernel4b<<< G1,B1>>>(c_targetImage,c_resultImage,c_probaJointHistogram,c_voxel_number,targetVoxelNumber,total_target_entries,activeVoxelNumber/* ,c_mask */);
	//reg_getJointHistogram_kernel5<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,targetVoxelNumber,total_target_entries,num_histogram_entries);
	//reg_getJointHistogram_kernel6<<< G1,B1,shared_size>>>(c_targetImage,c_resultImage,c_probaJointHistogram,targetVoxelNumber,total_target_entries,num_histogram_entries);
	NR_CUDA_CHECK_KERNEL(G1,B1);
	gettimeofday(&t4, NULL);

	NR_CUDA_SAFE_CALL((hipMemcpy(c_probaJointHistogram_int, c_probaJointHistogram, num_histogram_entries * sizeof(int), hipMemcpyDeviceToHost)));
	//NR_CUDA_SAFE_CALL((hipMemcpy(voxel_number_blk, c_voxel_number, reduce_size * sizeof(int), hipMemcpyDeviceToHost)));
	   
	
		for (i=0;i<num_histogram_entries;i++)
	{
		//printf("[NiftyReg Debug parag] index=%d probaJointHistogram= %d\n",i,c_probaJointHistogram_int[i]);
		probaJointHistogram[i]=(double)c_probaJointHistogram_int[i];
		voxel_number+=c_probaJointHistogram_int[i];
	}
			for (i=0;i<reduce_size;i++)
	{
		
		voxel_number+=voxel_number_blk[i];
		
	}	
	gettimeofday(&t2, NULL);
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;
	elapsedTime1 = (t4.tv_sec - t3.tv_sec) * 1000.0;      // sec to ms
    elapsedTime1 += (t4.tv_usec - t3.tv_usec) / 1000.0;
	elapsedTime2 = (t6.tv_sec - t5.tv_sec) * 1000.0;      // sec to ms
    elapsedTime2 += (t6.tv_usec - t5.tv_usec) / 1000.0;
	
	//printf("[NiftyReg F3D] Total joint hist filing in GPU=%f  and kernel time=%f copyHtoD=%f msec\n", elapsedTime,elapsedTime1,elapsedTime2 );
	//printf("[NiftyReg Debug parag] size=%lu\n",targetVoxelNumber * sizeof(float));
	//hipFree(c_mask);
	//hipFree(c_resultImage);
	NR_CUDA_SAFE_CALL(hipFree(c_probaJointHistogram));
	NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture));
/* 	hipHostFree(c_targetImage);
	hipHostFree(c_resultImage);
	hipHostFree(c_probaJointHistogram); */
	free(c_probaJointHistogram_int);
	free(voxel_number_blk);
	//fprintf(stderr,"[NiftyReg ERROR] The GPU implementation of new entropy calculation \n");
    

    int num_axes = num_target_volumes + num_result_volumes;
    if(approx || targetImage->nt>1 || resultImage->nt>1){
    // standard joint histogram filling has been used
    // Joint histogram has to be smoothed
        double window[3];
        window[0] = window[2] = GetBasisSplineValue((double)(-1.0));
        window[1] = GetBasisSplineValue((double)(0.0));

        double *histogram=NULL;
        double *result=NULL;

        // Smooth along each of the axes
        for (i = 0; i < num_axes; ++i)
        {
            // Use the arrays for storage of results
            if (i % 2 == 0) {
                result = logJointHistogram;
                histogram = probaJointHistogram;
            }
            else {
                result = probaJointHistogram;
                histogram = logJointHistogram;
            }
            traverse_and_smooth_axes(i, histogram, result, window,
                                             num_axes, histogram_dimensions);
        }

        // We may need to transfer the result
        if (result == logJointHistogram) memcpy(probaJointHistogram, logJointHistogram,
                                                sizeof(double)*num_probabilities);
    }// approx
	memset(logJointHistogram, 0, num_histogram_entries * sizeof(double));

    // Convert to probabilities
    for(i = 0; i < num_probabilities; ++i) {
        if (probaJointHistogram[i]) probaJointHistogram[i] /= voxel_number;
    }

    // Marginalise over all the result axes to generate the target entropy
    double *data = probaJointHistogram;
    double *store = logJointHistogram;
    double current_value, current_log;

    int count;
    double target_entropy = 0;
    {
        double scratch [num_probabilities/histogram_dimensions[num_axes - 1]];
        // marginalise over the result axes
        for (i = num_result_volumes-1, count = 0; i >= 0; --i, ++count)
        {
            traverse_and_sum_axes(num_axes - count - 1,
                                          data, store, num_axes - count,
                                          histogram_dimensions);

            if (count % 2 == 0) {
                data = logJointHistogram;
                store = scratch;
            }
            else {
                data = scratch;
                store = logJointHistogram;
            }
        }

        // Generate target entropy
        double *log_joint_target = &logJointHistogram[num_probabilities];

        for (i = 0; i < total_target_entries; ++i)
        {
            current_value = data[i];            
            current_log = 0;
            if (current_value) current_log = log(current_value);
            target_entropy -= current_value * current_log;
            log_joint_target[i] = current_log;
        }
    }
    memset(logJointHistogram, 0, num_probabilities * sizeof(double));
    data = probaJointHistogram;
    store = logJointHistogram;

    // Marginalise over the target axes
    double result_entropy = 0;
    {
        double scratch [num_probabilities / histogram_dimensions[0]];
        for (i = 0; i < num_target_volumes; ++i)
        {
            traverse_and_sum_axes(0, data, store, num_axes - i, &histogram_dimensions[i]);
            if (i % 2 == 0) {
                data = logJointHistogram;
                store = scratch;
            }
            else {
                data = scratch;
                store = logJointHistogram;
            }
        }
        // Generate result entropy
        double *log_joint_result = &logJointHistogram[num_probabilities+total_target_entries];

        for (i = 0; i < total_result_entries; ++i)
        {
            current_value = data[i];            
            current_log = 0;
            if (current_value) current_log = log(current_value);
            result_entropy -= current_value * current_log;
            log_joint_result[i] = current_log;
        }
    }

    // Generate joint entropy
    double joint_entropy = 0;
    for (i = 0; i < num_probabilities; ++i)
    {
        current_value = probaJointHistogram[i];        
        current_log = 0;
        if (current_value) current_log = log(current_value);
        joint_entropy -= current_value * current_log;
        logJointHistogram[i] = current_log;
    }

    entropies[0] = target_entropy;
    entropies[1] = result_entropy;
    entropies[2] = joint_entropy;
    entropies[3] = voxel_number;
/* 	printf("[NiftyReg Debug parag] entropies[0]=%f\n",entropies[0]);
	printf("[NiftyReg Debug parag] entropies[1]=%f\n",entropies[1]);
	printf("[NiftyReg Debug parag] entropies[2]=%f\n",entropies[2]);
	printf("[NiftyReg Debug parag] entropies[3]=%f\n",entropies[3]);
	exit(0); */
    return;

						  
}


/// Called when we have two target and two source image
void reg_getEntropies2x2_gpu(nifti_image *targetImages,
                             nifti_image *resultImages,
                             //int type,
                             unsigned int *target_bins, // should be an array of size num_target_volumes
                             unsigned int *result_bins, // should be an array of size num_result_volumes
                             double *probaJointHistogram,
                             double *logJointHistogram,
                             float  **logJointHistogram_d,
                             double *entropies,
                             int *mask)
{
    // The joint histogram is filled using the CPU arrays
    //Check the type of the target and source images
    if(targetImages->datatype!=NIFTI_TYPE_FLOAT32 || resultImages->datatype!=NIFTI_TYPE_FLOAT32){
        printf("[NiftyReg CUDA] reg_getEntropies2x2_gpu: This kernel should only be used floating images.\n");
        exit(1);
    }
    unsigned int voxelNumber = targetImages->nx*targetImages->ny*targetImages->nz;
    unsigned int binNumber = target_bins[0]*target_bins[1]*result_bins[0]*result_bins[1]+
                             target_bins[0]*target_bins[1]+result_bins[0]*result_bins[1];
    float *ref1Ptr = static_cast<float *>(targetImages->data);
    float *ref2Ptr = &ref1Ptr[voxelNumber];
    float *res1Ptr = static_cast<float *>(resultImages->data);
    float *res2Ptr = &res1Ptr[voxelNumber];
    int *maskPtr = &mask[0];
    memset(probaJointHistogram, 0, binNumber*sizeof(double));
    double voxelSum=0.;
    for(unsigned int i=0;i<voxelNumber;++i){
        if(*maskPtr++>-1){
            int val1 = static_cast<int>(*ref1Ptr);
            int val2 = static_cast<int>(*ref2Ptr);
            int val3 = static_cast<int>(*res1Ptr);
            int val4 = static_cast<int>(*res2Ptr);
            if(val1==val1 && val2==val2 && val3==val3 && val4==val4 &&
               val1>-1 && val1<(int)target_bins[0] && val2>-1 && val2<(int)target_bins[1] &&
               val3>-1 && val3<(int)result_bins[0] && val4>-1 && val4<(int)result_bins[1]){
                unsigned int index = ((val4*result_bins[0]+val3)*target_bins[1]+val2)*target_bins[0]+val1;
                probaJointHistogram[index]++;
                voxelSum++;
            }
        }
        ref1Ptr++;
        ref2Ptr++;
        res1Ptr++;
        res2Ptr++;
    }

    // The joint histogram is normalised and tranfered to the device
    float *logJointHistogram_float=NULL;
    NR_CUDA_SAFE_CALL(hipHostMalloc(&logJointHistogram_float,binNumber*sizeof(float)));
    for(unsigned int i=0;i<target_bins[0]*target_bins[1]*result_bins[0]*result_bins[1];++i)
        logJointHistogram_float[i]=float(probaJointHistogram[i]/voxelSum);

    NR_CUDA_SAFE_CALL(hipMemcpy(*logJointHistogram_d,logJointHistogram_float,binNumber*sizeof(float),hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipHostFree(logJointHistogram_float));

    float *tempHistogram=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&tempHistogram,binNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstTargetBin),&target_bins[0],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondTargetBin),&target_bins[1],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstResultBin),&result_bins[0],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondResultBin),&result_bins[1],sizeof(int)));


    // The joint histogram is smoothed along the x axis
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    dim3 B1(Block_reg_smoothJointHistogramX,1,1);
    const int gridSizesmoothJointHistogramX=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B1.x));
    dim3 G1(gridSizesmoothJointHistogramX,gridSizesmoothJointHistogramX,1);
    reg_smoothJointHistogramX_kernel <<< G1, B1 >>> (tempHistogram);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    // The joint histogram is smoothed along the y axis
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, tempHistogram, binNumber*sizeof(float)));
    dim3 B2(Block_reg_smoothJointHistogramY,1,1);
    const int gridSizesmoothJointHistogramY=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B2.x));
    dim3 G2(gridSizesmoothJointHistogramY,gridSizesmoothJointHistogramY,1);
    reg_smoothJointHistogramY_kernel <<< G2, B2 >>> (*logJointHistogram_d);
    NR_CUDA_CHECK_KERNEL(G2,B2)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    // The joint histogram is smoothed along the z axis
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    dim3 B3(Block_reg_smoothJointHistogramZ,1,1);
    const int gridSizesmoothJointHistogramZ=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B3.x));
    dim3 G3(gridSizesmoothJointHistogramZ,gridSizesmoothJointHistogramZ,1);
    reg_smoothJointHistogramZ_kernel <<< G3, B3 >>> (tempHistogram);
    NR_CUDA_CHECK_KERNEL(G3,B3)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    // The joint histogram is smoothed along the w axis
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, tempHistogram, binNumber*sizeof(float)));
    dim3 B4(Block_reg_smoothJointHistogramW,1,1);
    const int gridSizesmoothJointHistogramW=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B4.x));
    dim3 G4(gridSizesmoothJointHistogramW,gridSizesmoothJointHistogramW,1);
    reg_smoothJointHistogramW_kernel <<< G4, B4 >>> (*logJointHistogram_d);
    NR_CUDA_CHECK_KERNEL(G4,B4)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    NR_CUDA_SAFE_CALL(hipFree(tempHistogram));
    NR_CUDA_SAFE_CALL(hipHostMalloc(&logJointHistogram_float,binNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMemcpy(logJointHistogram_float,*logJointHistogram_d,binNumber*sizeof(float),hipMemcpyDeviceToHost));
    for(unsigned int i=0;i<target_bins[0]*target_bins[1]*result_bins[0]*result_bins[1];++i)
        probaJointHistogram[i]=logJointHistogram_float[i];
    NR_CUDA_SAFE_CALL(hipHostFree(logJointHistogram_float));

    // The 4D joint histogram is first marginalised along the x axis (target_bins[0])
    float *temp3DHistogram=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&temp3DHistogram,target_bins[1]*result_bins[0]*result_bins[1]*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    dim3 B5(Block_reg_marginaliseTargetX,1,1);
    const int gridSizesmoothJointHistogramA=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B5.x));
    dim3 G5(gridSizesmoothJointHistogramA,gridSizesmoothJointHistogramA,1);
    reg_marginaliseTargetX_kernel <<< G5, B5 >>> (temp3DHistogram);
    NR_CUDA_CHECK_KERNEL(G5,B5)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    // The 3D joint histogram is then marginalised along the y axis (target_bins[1])
    float *temp2DHistogram=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&temp2DHistogram,result_bins[0]*result_bins[1]*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, temp3DHistogram, target_bins[1]*result_bins[0]*result_bins[1]*sizeof(float)));
    dim3 B6(Block_reg_marginaliseTargetXY,1,1);
    const int gridSizesmoothJointHistogramB=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B6.x));
    dim3 G6(gridSizesmoothJointHistogramB,gridSizesmoothJointHistogramB,1);
    reg_marginaliseTargetXY_kernel <<< G6, B6 >>> (temp2DHistogram);
    NR_CUDA_CHECK_KERNEL(G6,B6)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));
    NR_CUDA_SAFE_CALL(hipFree(temp3DHistogram));

    // We need to transfer it to an array of floats (cannot directly copy it to probaJointHistogram
    // as that is an array of doubles) and hipMemcpy will produce unpredictable results
    const int total_target_entries = target_bins[0] * target_bins[1];
    const int total_result_entries = result_bins[0] * result_bins[1];
    const int num_probabilities =  total_target_entries * total_result_entries;
    int offset = num_probabilities + total_target_entries;    
    float *temp2DHistogram_h = new float[total_result_entries];
    hipMemcpy(temp2DHistogram_h,temp2DHistogram,total_result_entries*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < total_result_entries; ++i) {
        probaJointHistogram[offset + i] = temp2DHistogram_h[i];
    }
    delete[] temp2DHistogram_h;
    NR_CUDA_SAFE_CALL(hipFree(temp2DHistogram));


    // Now marginalise over the result axes.
    // First over W axes. (result_bins[1])
    temp3DHistogram=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&temp3DHistogram, target_bins[0]*target_bins[1]*result_bins[0]*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    dim3 B7(Block_reg_marginaliseResultX,1,1);
    const int gridSizesmoothJointHistogramC=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B7.x));
    dim3 G7(gridSizesmoothJointHistogramC,gridSizesmoothJointHistogramC,1);
    reg_marginaliseResultX_kernel <<< G7, B7 >>> (temp3DHistogram);
    NR_CUDA_CHECK_KERNEL(G7,B7)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    // Now over Z axes. (result_bins[0])
    temp2DHistogram=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&temp2DHistogram,target_bins[0]*target_bins[1]*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, temp3DHistogram, target_bins[0]*target_bins[1]*result_bins[0]*sizeof(float)));
    dim3 B8(Block_reg_marginaliseResultXY,1,1);
    const int gridSizesmoothJointHistogramD=(int)ceil(sqrtf((float)(target_bins[1]*result_bins[0]*result_bins[1])/(float)B8.x));
    dim3 G8(gridSizesmoothJointHistogramD,gridSizesmoothJointHistogramD,1);
    reg_marginaliseResultXY_kernel <<< G8, B8 >>> (temp2DHistogram);
    NR_CUDA_CHECK_KERNEL(G8,B8)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));

    hipFree(temp3DHistogram);
    // Transfer the data to CPU
    temp2DHistogram_h = new float[total_target_entries];
    hipMemcpy(temp2DHistogram_h,temp2DHistogram,total_target_entries*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < total_target_entries; ++i) {
        probaJointHistogram[num_probabilities + i] = temp2DHistogram_h[i];
    }    
    delete[] temp2DHistogram_h;
    hipFree(temp2DHistogram);

    // The next bits can be put on the GPU but there is not much performance gain and it is
    // better to go the log and accumulation using double precision.

    // Generate joint entropy
    float current_value, current_log;
    double joint_entropy = 0.0;
    for (int i = 0; i < num_probabilities; ++i)
    {
        current_value = probaJointHistogram[i];
        current_log = 0.0;
        if (current_value) current_log = log(current_value);
        joint_entropy -= current_value * current_log;
        logJointHistogram[i] = current_log;
    }

    // Generate target entropy
    double *log_joint_target = &logJointHistogram[num_probabilities];
    double target_entropy = 0.0;
    for (int i = 0; i < total_target_entries; ++i)
    {
        current_value = probaJointHistogram[num_probabilities + i];
        current_log = 0.0;
        if (current_value) current_log = log(current_value);
        target_entropy -= current_value * current_log;
        log_joint_target[i] = current_log;
    }

    // Generate result entropy
    double *log_joint_result = &logJointHistogram[num_probabilities+total_target_entries];
    double result_entropy = 0.0;
    for (int i = 0; i < total_result_entries; ++i)
    {
        current_value = probaJointHistogram[num_probabilities + total_target_entries + i];
        current_log = 0.0;
        if (current_value) current_log = log(current_value);
        result_entropy -= current_value * current_log;
        log_joint_result[i] = current_log;
    }

    entropies[0] = target_entropy;
    entropies[1] = result_entropy;
    entropies[2] = joint_entropy;
    entropies[3] = voxelSum;
}

/// Called when we only have one target and one source image
void reg_getVoxelBasedNMIGradientUsingPW_gpu(   nifti_image *targetImage,
                                                nifti_image *resultImage,
                                                hipArray **targetImageArray_d,
                                                float **resultImageArray_d,
                                                float4 **resultGradientArray_d,
                                                float **logJointHistogram_d,
                                                float4 **voxelNMIGradientArray_d,
                                                int **mask_d,
                                                int activeVoxelNumber,
                                                double *entropies,
                                                int refBinning,
                                                int floBinning)
{
    if(resultImage!=resultImage)
        printf("Useless lines to avoid a warning");

    const int voxelNumber = targetImage->nx*targetImage->ny*targetImage->nz;
    const int3 imageSize=make_int3(targetImage->nx,targetImage->ny,targetImage->nz);
    const int binNumber = refBinning*floBinning+refBinning+floBinning;
    const float4 entropies_h=make_float4((float)entropies[0],(float)entropies[1],(float)entropies[2],(float)entropies[3]);
    const float NMI = (float)((entropies[0]+entropies[1])/entropies[2]);

    // Bind Symbols
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize),&imageSize,sizeof(int3)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstTargetBin),&refBinning,sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstResultBin),&floBinning,sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Entropies),&entropies_h,sizeof(float4)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NMI),&NMI,sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)));

    // Texture bindingcurrentFloating
    //Bind target image array to a 3D texture
    firstTargetImageTexture.normalized = true;
    firstTargetImageTexture.filterMode = hipFilterModeLinear;
    firstTargetImageTexture.addressMode[0] = hipAddressModeWrap;
    firstTargetImageTexture.addressMode[1] = hipAddressModeWrap;
    firstTargetImageTexture.addressMode[2] = hipAddressModeWrap;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(firstTargetImageTexture, *targetImageArray_d, channelDesc))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageTexture, *resultImageArray_d, voxelNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageGradientTexture, *resultGradientArray_d, voxelNumber*sizeof(float4)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemset(*voxelNMIGradientArray_d, 0, voxelNumber*sizeof(float4)));

    const unsigned int Grid_reg_getVoxelBasedNMIGradientUsingPW =
        (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_getVoxelBasedNMIGradientUsingPW));
    dim3 B1(Block_reg_getVoxelBasedNMIGradientUsingPW,1,1);
    dim3 G1(Grid_reg_getVoxelBasedNMIGradientUsingPW,Grid_reg_getVoxelBasedNMIGradientUsingPW,1);

    reg_getVoxelBasedNMIGradientUsingPW_kernel <<< G1, B1 >>> (*voxelNMIGradientArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstTargetImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstResultImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstResultImageGradientTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture));
}
/// Called when we have two target and two source image
void reg_getVoxelBasedNMIGradientUsingPW2x2_gpu(nifti_image *targetImage,
                                                nifti_image *resultImage,
                                                hipArray **targetImageArray1_d,
                                                hipArray **targetImageArray2_d,
                                                float **resultImageArray1_d,
                                                float **resultImageArray2_d,
                                                float4 **resultGradientArray1_d,
                                                float4 **resultGradientArray2_d,
                                                float **logJointHistogram_d,
                                                float4 **voxelNMIGradientArray_d,
                                                int **mask_d,
                                                int activeVoxelNumber,
                                                double *entropies,
                                                unsigned int *targetBinning,
                                                unsigned int *resultBinning)
{
    if (targetImage->nt != 2 || resultImage->nt != 2) {
        printf("[NiftyReg CUDA] reg_getVoxelBasedNMIGradientUsingPW2x2_gpu: This kernel should only be used with two target and source images\n");
        return;
    }
    const int voxelNumber = targetImage->nx*targetImage->ny*targetImage->nz;
    const int3 imageSize=make_int3(targetImage->nx,targetImage->ny,targetImage->nz);
    const float4 entropies_h=make_float4((float)entropies[0],(float)entropies[1],(float)entropies[2],(float)entropies[3]);
    const float NMI = (float)((entropies[0]+entropies[1])/entropies[2]);
    const int binNumber = targetBinning[0]*targetBinning[1]*resultBinning[0]*resultBinning[1] + (targetBinning[0]*targetBinning[1]) + (resultBinning[0]*resultBinning[1]);

    // Bind Symbols
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&voxelNumber,sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize),&imageSize,sizeof(int3)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstTargetBin),&targetBinning[0],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondTargetBin),&targetBinning[1],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_firstResultBin),&resultBinning[0],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_secondResultBin),&resultBinning[1],sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Entropies),&entropies_h,sizeof(float4)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NMI),&NMI,sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)));

    // Texture binding
    firstTargetImageTexture.normalized = true;
    firstTargetImageTexture.filterMode = hipFilterModeLinear;
    firstTargetImageTexture.addressMode[0] = hipAddressModeWrap;
    firstTargetImageTexture.addressMode[1] = hipAddressModeWrap;
    firstTargetImageTexture.addressMode[2] = hipAddressModeWrap;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(firstTargetImageTexture, *targetImageArray1_d, channelDesc))
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(secondTargetImageTexture, *targetImageArray2_d, channelDesc))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageTexture, *resultImageArray1_d, voxelNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, secondResultImageTexture, *resultImageArray2_d, voxelNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, firstResultImageGradientTexture, *resultGradientArray1_d, voxelNumber*sizeof(float4)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, secondResultImageGradientTexture, *resultGradientArray2_d, voxelNumber*sizeof(float4)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, histogramTexture, *logJointHistogram_d, binNumber*sizeof(float)));
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMemset(*voxelNMIGradientArray_d, 0, voxelNumber*sizeof(float4)));

    const unsigned int Grid_reg_getVoxelBasedNMIGradientUsingPW2x2 =
        (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_getVoxelBasedNMIGradientUsingPW2x2));
    dim3 B1(Block_reg_getVoxelBasedNMIGradientUsingPW2x2,1,1);
    dim3 G1(Grid_reg_getVoxelBasedNMIGradientUsingPW2x2,Grid_reg_getVoxelBasedNMIGradientUsingPW2x2,1);

    reg_getVoxelBasedNMIGradientUsingPW2x2_kernel <<< G1, B1 >>> (*voxelNMIGradientArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstTargetImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(secondTargetImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstResultImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(secondResultImageTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(firstResultImageGradientTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(secondResultImageGradientTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(histogramTexture));
    NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture));
}

#endif
