#include "hip/hip_runtime.h"
/*
 *  _reg_resampling_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_RESAMPLING_GPU_CU
#define _REG_RESAMPLING_GPU_CU

#include "_reg_resampling_gpu.h"
#include "_reg_resampling_kernels.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <unistd.h>
#include <sys/time.h>

/* *************************************************************** */
/* *************************************************************** */
void reg_resampleSourceImage_gpu(nifti_image *sourceImage,
                                float **resultImageArray_d,
                                hipArray **sourceImageArray_d,
                                float4 **positionFieldImageArray_d,
                                int **mask_d,
                                int activeVoxelNumber,
                                float sourceBGValue)
{
    int3 sourceDim = make_int3(sourceImage->nx, sourceImage->ny, sourceImage->nz);

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_SourceDim),&sourceDim,sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddingValue),&sourceBGValue,sizeof(float)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)))
	struct timeval t1, t2;
    double elapsedTime;
	
    //Bind source image array to a 3D texture
    sourceTexture.normalized = true;
    sourceTexture.filterMode = hipFilterModeLinear;
    sourceTexture.addressMode[0] = hipAddressModeWrap;
    sourceTexture.addressMode[1] = hipAddressModeWrap;
    sourceTexture.addressMode[2] = hipAddressModeWrap;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(sourceTexture, *sourceImageArray_d, channelDesc))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, positionFieldTexture, *positionFieldImageArray_d, activeVoxelNumber*sizeof(float4)))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)))

    // Bind the real to voxel matrix to texture
    mat44 *sourceMatrix;
    if(sourceImage->sform_code>0)
        sourceMatrix=&(sourceImage->sto_ijk);
    else sourceMatrix=&(sourceImage->qto_ijk);
    float4 *sourceRealToVoxel_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sourceRealToVoxel_h, 3*sizeof(float4)))
    float4 *sourceRealToVoxel_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sourceRealToVoxel_d, 3*sizeof(float4)))
    for(int i=0; i<3; i++){
        sourceRealToVoxel_h[i].x=sourceMatrix->m[i][0];
        sourceRealToVoxel_h[i].y=sourceMatrix->m[i][1];
        sourceRealToVoxel_h[i].z=sourceMatrix->m[i][2];
        sourceRealToVoxel_h[i].w=sourceMatrix->m[i][3];
    }
	gettimeofday(&t1, NULL);
    NR_CUDA_SAFE_CALL(hipMemcpy(sourceRealToVoxel_d, sourceRealToVoxel_h, 3*sizeof(float4), hipMemcpyHostToDevice))
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sourceRealToVoxel_h))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, sourceMatrixTexture, sourceRealToVoxel_d, 3*sizeof(float4)))

    const unsigned int Grid_reg_resampleSourceImage = (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_resampleSourceImage));
    dim3 B1(Block_reg_resampleSourceImage,1,1);
    dim3 G1(Grid_reg_resampleSourceImage,Grid_reg_resampleSourceImage,1);
    reg_resampleSourceImage_kernel <<< G1, B1 >>> (*resultImageArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(positionFieldTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceMatrixTexture))

    hipFree(sourceRealToVoxel_d);
	gettimeofday(&t2, NULL);
	elapsedTime = (t2.tv_sec - t1.tv_sec) * 1000.0;      // sec to ms
    elapsedTime += (t2.tv_usec - t1.tv_usec) / 1000.0;
	printf("[NiftyReg F3D] reg_resampleSourceImage_kernel time =%f msec\n", elapsedTime);
	//printf("[NiftyReg F3D] reg_resampleSourceImage_kernel throughput =%f voxel per sec\n", (activeVoxelNumber*1000)/elapsedTime);
}
/* *************************************************************** */
/* *************************************************************** */
void reg_getSourceImageGradient_gpu(nifti_image *sourceImage,
                                    hipArray **sourceImageArray_d,
                                    float4 **positionFieldImageArray_d,
                                    float4 **resultGradientArray_d,
                                    int activeVoxelNumber,
									int *mask_d)
{
    int3 sourceDim = make_int3(sourceImage->nx, sourceImage->ny, sourceImage->nz);
	//printf("activeVoxelNumber=%d\n",activeVoxelNumber);

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_SourceDim), &sourceDim, sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber), &activeVoxelNumber, sizeof(int)))

    //Bind source image array to a 3D texture
    sourceTexture.normalized = true;
    sourceTexture.filterMode = hipFilterModeLinear;
    sourceTexture.addressMode[0] = hipAddressModeWrap;
    sourceTexture.addressMode[1] = hipAddressModeWrap;
    sourceTexture.addressMode[2] = hipAddressModeWrap;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    NR_CUDA_SAFE_CALL(hipBindTextureToArray(sourceTexture, *sourceImageArray_d, channelDesc))
	NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, mask_d, activeVoxelNumber*sizeof(int)))

    //Bind positionField to texture
    NR_CUDA_SAFE_CALL(hipBindTexture(0, positionFieldTexture, *positionFieldImageArray_d, activeVoxelNumber*sizeof(float4)))

    // Bind the real to voxel matrix to texture
    mat44 *sourceMatrix;
    if(sourceImage->sform_code>0)
        sourceMatrix=&(sourceImage->sto_ijk);
    else sourceMatrix=&(sourceImage->qto_ijk);
    float4 *sourceRealToVoxel_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sourceRealToVoxel_h, 3*sizeof(float4)))
    float4 *sourceRealToVoxel_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sourceRealToVoxel_d, 3*sizeof(float4)))
    for(int i=0; i<3; i++){
        sourceRealToVoxel_h[i].x=sourceMatrix->m[i][0];
        sourceRealToVoxel_h[i].y=sourceMatrix->m[i][1];
        sourceRealToVoxel_h[i].z=sourceMatrix->m[i][2];
        sourceRealToVoxel_h[i].w=sourceMatrix->m[i][3];
    }
    NR_CUDA_SAFE_CALL(hipMemcpy(sourceRealToVoxel_d, sourceRealToVoxel_h, 3*sizeof(float4), hipMemcpyHostToDevice))
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sourceRealToVoxel_h))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, sourceMatrixTexture, sourceRealToVoxel_d, 3*sizeof(float4)))

    const unsigned int Grid_reg_getSourceImageGradient = (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)Block_reg_getSourceImageGradient));
    dim3 B1(Block_reg_getSourceImageGradient,1,1);
    dim3 G1(Grid_reg_getSourceImageGradient,Grid_reg_getSourceImageGradient,1);
    reg_getSourceImageGradient_kernel <<< G1, B1 >>> (*resultGradientArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(positionFieldTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(sourceMatrixTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture))

    hipFree(sourceRealToVoxel_d);
}
/* *************************************************************** */
/* *************************************************************** */
void reg_randomsamplingMask_gpu(int *mask_d,
							int samples,
							int activeVoxelNumber)
{	dim3 B1(1024,1,1);
    dim3 G1(samples/1024,1,1);
	
	  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
	NR_CUDA_SAFE_CALL(hipMalloc((void**) &states, samples * sizeof(hiprandState_t)))
  
	//init<<<G1, B1 >>>(time(0), states);
	
	
	NR_CUDA_SAFE_CALL(hipMalloc(&mask_d,samples*sizeof(int)))
	reg_randomsamplingMask_kernel <<< G1, B1 >>> (states,mask_d,samples,activeVoxelNumber,time(0));
	NR_CUDA_CHECK_KERNEL(G1,B1)
	
	//random sampling checking
/* 	int *targetMask_h; 
	targetMask_h=(int *)malloc(samples * sizeof(int));
	NR_CUDA_SAFE_CALL(hipMemcpy(targetMask_h,mask_d,samples*sizeof(int),hipMemcpyDeviceToHost))
	for (int i=0;i<samples;i++)
	{
		printf("targetMask_h[%d]=%d\n",i,targetMask_h[i]);
	}
	free(targetMask_h); */
	
}

#endif
