#include "hip/hip_runtime.h"
/*
 *  _reg_mutualinformation_kernels.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_MUTUALINFORMATION_kernels_CU
#define _REG_MUTUALINFORMATION_kernels_CU

#include <stdio.h>

#define COEFF_L 0.16666666f
#define COEFF_C 0.66666666f
#define COEFF_B 0.83333333f

__device__ __constant__ int c_VoxelNumber;
__device__ __constant__ int3 c_ImageSize;

// Bins: Need 4 values for max 4 channels.
__device__ __constant__ int c_firstTargetBin;
__device__ __constant__ int c_secondTargetBin;
__device__ __constant__ int c_firstResultBin;
__device__ __constant__ int c_secondResultBin;

__device__ __constant__ float4 c_Entropies;
__device__ __constant__ float c_NMI;
__device__ __constant__ int c_ActiveVoxelNumber;

texture<float, 3, hipReadModeElementType> firstTargetImageTexture;
texture<float, 1, hipReadModeElementType> firstResultImageTexture;
texture<float4, 1, hipReadModeElementType> firstResultImageGradientTexture;
texture<float, 1, hipReadModeElementType> histogramTexture;
texture<float4, 1, hipReadModeElementType> gradientImageTexture;
texture<int, 1, hipReadModeElementType> maskTexture;

/// Added for the multichannel stuff. We currently only support 2 target and 2 source channels.
/// So we need another texture for the second target and source channel respectively.
texture<float, 3, hipReadModeElementType> secondTargetImageTexture;
texture<float, 1, hipReadModeElementType> secondResultImageTexture;
texture<float4, 1, hipReadModeElementType> secondResultImageGradientTexture;

__global__ void reg_getJointHistogram_kernel1(float *targetImage, float *resultImage,int *probaJointHistogram,int *c_voxel_number,int targetVoxelNumber,int total_target_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	extern __shared__ int sdata[];
	unsigned int td = threadIdx.x;
	unsigned int value=0;
	
	if (tid < targetVoxelNumber)
	{
		//unsigned int td = threadIdx.x;
	  float target_values=targetImage[tid];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[tid];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		
			//atomicAdd(&probaJointHistogram[int(round(target_values))+int(round(result_values))*total_target_entries],1);
			//atomicAdd(&probaJointHistogram[(__float2int_rd(target_values))+(__float2int_rd(result_values))*68],1); // lot diff
			//atomicAdd(&probaJointHistogram[(__float2int_ru(target_values))+(__float2int_ru(result_values))*total_target_entries],1);
			atomicAdd(&probaJointHistogram[(__float2int_rn(target_values))+(__float2int_rn(result_values))*total_target_entries],1);
			value=1;
		  //atomicAdd(&added_value,1);
		  //printf("[kernel debug] index=%d value=%d\n",tid,__float2int_ru(round(target_values))+__float2int_ru(round(result_values)));
		  
	  }
	  //sdata[td]=value;

	}
		  	sdata[td]=value;
			//printf("sdata[td]=%d\n",value);
			__syncthreads();
			for(unsigned int s=1; s < blockDim.x; s *= 2) 
			{
			if (td % (2*s) == 0)
			{
				sdata[td] += sdata[td + s];
			}
				__syncthreads();
		    }
		if (td == 0) 
		{
		  c_voxel_number[blockIdx.x]=sdata[0];
		  //printf("zero=%d",td);
		}
	return;
} 
 __global__ void reg_getJointHistogram_kernel2(float *targetImage, float *resultImage,int *probaJointHistogram,int *c_voxel_number,int targetVoxelNumber,int total_target_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	extern __shared__ int sdata[];
	unsigned int td = threadIdx.x;
	unsigned int value=0;
	
	
	if (tid < targetVoxelNumber)
	{
		//unsigned int td = threadIdx.x;
	  float target_values=targetImage[tid];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[tid];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		
			//atomicAdd(&probaJointHistogram[int(round(target_values))+int(round(result_values))*total_target_entries],1);
			//atomicAdd(&probaJointHistogram[(__float2int_rd(target_values))+(__float2int_rd(result_values))*68],1); // lot diff
			//atomicAdd(&probaJointHistogram[(__float2int_ru(target_values))+(__float2int_ru(result_values))*total_target_entries],1);
			atomicAdd(&probaJointHistogram[(__float2int_rn(target_values))+(__float2int_rn(result_values))*total_target_entries],1);
			value=1;
		  //atomicAdd(&added_value,1);
		  //printf("[kernel debug] index=%d value=%d\n",tid,__float2int_ru(round(target_values))+__float2int_ru(round(result_values)));
		  
	  }
	  //sdata[td]=value;

	}
		  	sdata[td]=value;
			//printf("sdata[td]=%d\n",value);
			__syncthreads();
			for(unsigned int s=1; s < blockDim.x; s *= 2) 
			{
				int index = 2 * s * td;
			if (index < blockDim.x )
			{
				sdata[index] += sdata[index + s];
			}
				__syncthreads();
		    }
		if (td == 0) 
		{
		  c_voxel_number[blockIdx.x]=sdata[0];
		  //printf("zero=%d",td);
		}
	return;
} 

__global__ void reg_getJointHistogram_kernel3(float *targetImage, float *resultImage,int *probaJointHistogram,int *c_voxel_number,int targetVoxelNumber,int total_target_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	extern __shared__ int sdata[];
	unsigned int td = threadIdx.x;
	unsigned int value=0;
	
	
	if (tid < targetVoxelNumber)
	{
		//unsigned int td = threadIdx.x;
	  float target_values=targetImage[tid];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[tid];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		
			//atomicAdd(&probaJointHistogram[int(round(target_values))+int(round(result_values))*total_target_entries],1);
			//atomicAdd(&probaJointHistogram[(__float2int_rd(target_values))+(__float2int_rd(result_values))*68],1); // lot diff
			//atomicAdd(&probaJointHistogram[(__float2int_ru(target_values))+(__float2int_ru(result_values))*total_target_entries],1);
			atomicAdd(&probaJointHistogram[(__float2int_rn(target_values))+(__float2int_rn(result_values))*total_target_entries],1);
			value=1;
		  //atomicAdd(&added_value,1);
		  //printf("[kernel debug] index=%d value=%d\n",tid,__float2int_ru(round(target_values))+__float2int_ru(round(result_values)));
		  
	  }
	  //sdata[td]=value;

	}
		  	sdata[td]=value;
			//printf("sdata[td]=%d\n",value);
			__syncthreads();
			for (unsigned int s=blockDim.x/2; s>0; s>>=1)
				{
				if (td < s) {
					sdata[td] += sdata[td + s];
							}
					__syncthreads();
				}
		if (td == 0) 
		{
		  c_voxel_number[blockIdx.x]=sdata[0];
		  //printf("zero=%d",td);
		}
	return;
} 

__global__ void reg_getJointHistogram_kernel4(float *targetImage, float *resultImage,int *probaJointHistogram,int *c_voxel_number,int targetVoxelNumber,int total_target_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	extern __shared__ int sdata[];
	unsigned int td = threadIdx.x;
	unsigned int value=0;
	
	
	if (tid < targetVoxelNumber)
	{
		//unsigned int td = threadIdx.x;
	  float target_values=targetImage[tid];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[tid];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		
	
			atomicAdd(&probaJointHistogram[(__float2int_rn(target_values))+(__float2int_rn(result_values))*total_target_entries],1);
			value=1;

		  
	  }
	 

	}
		  	sdata[td]=value;
			//printf("sdata[td]=%d\n",value);
			__syncthreads();
		for (unsigned int s=blockDim.x/2; s>32; s>>=1)
		{
		if (td < s)
		{sdata[td] += sdata[td + s];
			
		}
		__syncthreads();
		}
		if (td < 32)
			{
				sdata[td] += sdata[td + 32];__syncthreads();
				sdata[td] += sdata[td + 16];__syncthreads();
				sdata[td] += sdata[td + 8];__syncthreads();
				sdata[td] += sdata[td + 4];__syncthreads();
				sdata[td] += sdata[td + 2];__syncthreads();
				sdata[td] += sdata[td + 1];__syncthreads();
			}
			//__syncthreads();
				if (td == 0) 
		{
		  c_voxel_number[blockIdx.x]=sdata[0];
		  //printf("zero=%d",td);
		}
	return;
} 
//template <unsigned int blockSize>
__global__ void reg_getJointHistogram_kernel5(float *targetImage, float *resultImage,int *probaJointHistogram,int targetVoxelNumber,int total_target_entries,int num_histogram_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	//extern __shared__ int sdata[];
	extern __shared__ int local_hist[];
	//unsigned int td = threadIdx.x;
	//unsigned int value=0;
	int hist_index=0;
	for ( int i = threadIdx.x; i < num_histogram_entries; i += blockDim.x ) {
			local_hist[i] = 0;
		}
	if (tid < targetVoxelNumber)
	{
		//unsigned int td = threadIdx.x;
	/* 	if (tid < num_histogram_entries )
		{
			local_hist[tid] = 0;
		}
		__syncthreads(); */
		
	  float target_values=targetImage[tid];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[tid];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		

			hist_index=__float2int_rn(target_values)+__float2int_rn(result_values)*total_target_entries;
			atomicAdd(&local_hist[hist_index],1);
			//value=1;

		  
	  }
/* 	  __syncthreads();
					if (tid < num_histogram_entries )
		{
			atomicAdd(&probaJointHistogram[tid],local_hist[tid]);
		} */

	}
		__syncthreads();
		for ( int i = threadIdx.x; i < num_histogram_entries; i += blockDim.x ) {
			atomicAdd(&probaJointHistogram[i],local_hist[i]);
		}  	
	return;
}  

__global__ void reg_getJointHistogram_kernel6(float *targetImage, float *resultImage,int *probaJointHistogram,int targetVoxelNumber,int total_target_entries, int num_histogram_entries)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	
	//extern __shared__ int sdata[];
	extern __shared__ int local_hist[];
	//unsigned int td = threadIdx.x;
	//unsigned int value=0;
	int hist_index=0;
	for ( int i = threadIdx.x; i < num_histogram_entries; i += blockDim.x ) {
			local_hist[i] = 0;
		}
		
	__syncthreads();
	
	for (int i=tid;i<targetVoxelNumber;i+=blockDim.x )
	{
			
	  float target_values=targetImage[i];
	  
	  float result_values;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	 result_values=resultImage[i];
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		

			hist_index=__float2int_rn(target_values)+__float2int_rn(result_values)*total_target_entries;
			atomicAdd(&local_hist[hist_index],1);
			//value=1;

		  
	  }
		
		
	}
	

	__syncthreads();
		for ( int j = threadIdx.x; j < num_histogram_entries; j += blockDim.x ) {
			atomicAdd(&probaJointHistogram[j],local_hist[j]);
		}  	
	return;
}  


__global__ void reg_getJointHistogram_kernel4b(float *targetImage, float *resultImage,int *probaJointHistogram,int *c_voxel_number,int targetVoxelNumber,int total_target_entries,int activeVoxelNumber/* , int *mask */)
{
	const int tid= (blockIdx.x)*blockDim.x+threadIdx.x;
	//extern __shared__ int sdata[];
	//unsigned int td = threadIdx.x;
	//unsigned int value=0;
	//printf("here \n");
	
	if (tid < activeVoxelNumber)
	{	
		//const int v_mask=mask[tid];
		const int index = tex1Dfetch(maskTexture,tid);
		//const int v_mask = tex1D(maskTexture,tid);
		/* if (tid==10)
		{
			printf("mask[%d]=%d\n",tid,v_mask);
		}
		if (v_mask > -1)
		{ */
		//printf("mask[%d]=%d\n",tid,v_mask);
		//unsigned int td = threadIdx.x;
	  const float target_values=targetImage[index];
	  
	  const float result_values=resultImage[index];;
	  bool valid_values = true;
	  //__shared__ int added_value;
	                if (target_values < 0 || target_values >= total_target_entries || target_values != target_values) 
					{
                    valid_values = false;
                    }
	 if (valid_values)
	 {	// result_values=
		
		if (result_values <  0 ||
                    result_values >= total_target_entries ||
                    result_values != result_values) {
                    valid_values = false;
					
					
              }
		 }
	  if (valid_values)
	  {		
	
			atomicAdd(&probaJointHistogram[(__float2int_rn(target_values))+(__float2int_rn(result_values))*total_target_entries],1);
			//value=1;

		  
	  }
	 
		//}
	}
		  	
	return;
} 



__device__ float GetBasisSplineValue(float x)
{
    x=fabsf(x);
    float value=0.0f;
    if(x<2.0f)
        if(x<1.0f)
            value = 2.0f/3.0f + (0.5f*x-1.0f)*x*x;
        else{
            x-=2.0f;
            value = -x*x*x/6.0f;
    }
    return value;
}
__device__ float GetBasisSplineDerivativeValue(float ori)
{
    float x=fabsf(ori);
    float value=0.0f;
    if(x<2.0f)
        if(x<1.0f)
            value = (1.5f*x-2.0f)*ori;
        else{
            x-=2.0f;
            value = -0.5f * x * x;
            if(ori<0.0f)value =-value;
    }
    return value;
}

__global__ void reg_getVoxelBasedNMIGradientUsingPW_kernel(float4 *voxelNMIGradientArray_d)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_ActiveVoxelNumber){

        const int targetIndex = tex1Dfetch(maskTexture,tid);
        int tempIndex=targetIndex;
        const int z = tempIndex/(c_ImageSize.x*c_ImageSize.y);
        tempIndex  -= z*c_ImageSize.x*c_ImageSize.y;
        const int y = tempIndex/c_ImageSize.x;
        const int x = tempIndex - y*c_ImageSize.x;

        float targetImageValue = tex3D(firstTargetImageTexture,
                                       ((float)x+0.5f)/(float)c_ImageSize.x,
                                       ((float)y+0.5f)/(float)c_ImageSize.y,
                                       ((float)z+0.5f)/(float)c_ImageSize.z);
        float resultImageValue = tex1Dfetch(firstResultImageTexture,targetIndex);
        float4 resultImageGradient = tex1Dfetch(firstResultImageGradientTexture,tid);

        float4 gradValue = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        // No computation is performed if any of the point is part of the background
        // The two is added because the image is resample between 2 and bin +2
        // if 64 bins are used the histogram will have 68 bins et the image will be between 2 and 65
        if( targetImageValue>0.0f &&
            resultImageValue>0.0f &&
            targetImageValue<c_firstTargetBin &&
            resultImageValue<c_firstResultBin &&
            targetImageValue==targetImageValue &&
            resultImageValue==resultImageValue){

            targetImageValue = floor(targetImageValue);
            resultImageValue = floor(resultImageValue);

            float3 resDeriv = make_float3(
                resultImageGradient.x,
                resultImageGradient.y,
                resultImageGradient.z);

            if( resultImageGradient.x==resultImageGradient.x &&
                resultImageGradient.y==resultImageGradient.y &&
                resultImageGradient.z==resultImageGradient.z){

                float jointEntropyDerivative_X = 0.0f;
                float movingEntropyDerivative_X = 0.0f;
                float fixedEntropyDerivative_X = 0.0f;

                float jointEntropyDerivative_Y = 0.0f;
                float movingEntropyDerivative_Y = 0.0f;
                float fixedEntropyDerivative_Y = 0.0f;

                float jointEntropyDerivative_Z = 0.0f;
                float movingEntropyDerivative_Z = 0.0f;
                float fixedEntropyDerivative_Z = 0.0f;

                for(int t=(int)(targetImageValue-1.0f); t<(int)(targetImageValue+2.0f); t++){
                    if(-1<t && t<c_firstTargetBin){
                        for(int r=(int)(resultImageValue-1.0f); r<(int)(resultImageValue+2.0f); r++){
                            if(-1<r && r<c_firstResultBin){
                                float commonValue = GetBasisSplineValue((float)t-targetImageValue) *
                                    GetBasisSplineDerivativeValue((float)r-resultImageValue);

                                float jointLog = tex1Dfetch(histogramTexture, r*c_firstResultBin+t);
                                float targetLog = tex1Dfetch(histogramTexture, c_firstTargetBin*c_firstResultBin+t);
                                float resultLog = tex1Dfetch(histogramTexture, c_firstTargetBin*c_firstResultBin+c_firstTargetBin+r);

                                float temp = commonValue * resDeriv.x;
                                jointEntropyDerivative_X -= temp * jointLog;
                                fixedEntropyDerivative_X -= temp * targetLog;
                                movingEntropyDerivative_X -= temp * resultLog;

                                temp = commonValue * resDeriv.y;
                                jointEntropyDerivative_Y -= temp * jointLog;
                                fixedEntropyDerivative_Y -= temp * targetLog;
                                movingEntropyDerivative_Y -= temp * resultLog;

                                temp = commonValue * resDeriv.z;
                                jointEntropyDerivative_Z -= temp * jointLog;
                                fixedEntropyDerivative_Z -= temp * targetLog;
                                movingEntropyDerivative_Z -= temp * resultLog;
                            } // O<t<bin
                        } // t
                    } // 0<r<bin
                } // r

                float NMI= c_NMI;
                float temp = c_Entropies.z;
                // (Marc) I removed the normalisation by the voxel number as each gradient has to be normalised in the same way
                gradValue.x = (fixedEntropyDerivative_X + movingEntropyDerivative_X - NMI * jointEntropyDerivative_X) / temp;
                gradValue.y = (fixedEntropyDerivative_Y + movingEntropyDerivative_Y - NMI * jointEntropyDerivative_Y) / temp;
                gradValue.z = (fixedEntropyDerivative_Z + movingEntropyDerivative_Z - NMI * jointEntropyDerivative_Z) / temp;

            }
        }
        voxelNMIGradientArray_d[targetIndex]=gradValue;

    }
    return;
}

// Multichannel NMI gradient. Hardcoded for 2x2 NMI channels.
__global__ void reg_getVoxelBasedNMIGradientUsingPW2x2_kernel(float4 *voxelNMIGradientArray_d)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_ActiveVoxelNumber){
        const int targetIndex = tex1Dfetch(maskTexture,tid);
        int tempIndex=targetIndex;
        const int z = tempIndex/(c_ImageSize.x*c_ImageSize.y);
        tempIndex  -= z*c_ImageSize.x*c_ImageSize.y;
        const int y = tempIndex/c_ImageSize.x;
        const int x = tempIndex - y*c_ImageSize.x;

        float4 voxelValues = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        voxelValues.x = tex3D(firstTargetImageTexture,
                              ((float)x+0.5f)/(float)c_ImageSize.x,
                              ((float)y+0.5f)/(float)c_ImageSize.y,
                              ((float)z+0.5f)/(float)c_ImageSize.z);
        voxelValues.x = tex3D(secondTargetImageTexture,
                              ((float)x+0.5f)/(float)c_ImageSize.x,
                              ((float)y+0.5f)/(float)c_ImageSize.y,
                              ((float)z+0.5f)/(float)c_ImageSize.z);
        voxelValues.z = tex1Dfetch(firstResultImageTexture,targetIndex);
        voxelValues.w = tex1Dfetch(secondResultImageTexture,targetIndex);

        float4 firstResultImageGradient = tex1Dfetch(firstResultImageGradientTexture,tid);
        float4 secondResultImageGradient = tex1Dfetch(secondResultImageGradientTexture,tid);
        float4 gradValue = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        // Could remove some tests (which are not really needed) to reduce register
        // count. They should be put in again at some point for completeness and generality.
        if (voxelValues.x == voxelValues.x &&
            voxelValues.y == voxelValues.y &&
            voxelValues.z == voxelValues.z &&
            voxelValues.w == voxelValues.w &&
            voxelValues.x >= 0.0f &&
            voxelValues.y >= 0.0f &&
            voxelValues.z >= 0.0f &&
            voxelValues.w >= 0.0f &&
            voxelValues.x < c_firstTargetBin &&
            voxelValues.y < c_secondTargetBin &&
            voxelValues.z < c_firstResultBin &&
            voxelValues.w < c_secondResultBin)
        {
            voxelValues.x = (float)((int)voxelValues.x);
            voxelValues.y = (float)((int)voxelValues.y);
            voxelValues.z = (float)((int)voxelValues.z);
            voxelValues.w = (float)((int)voxelValues.w);

            if( firstResultImageGradient.x==firstResultImageGradient.x &&
                firstResultImageGradient.y==firstResultImageGradient.y &&
                firstResultImageGradient.z==firstResultImageGradient.z &&
                secondResultImageGradient.x==secondResultImageGradient.x &&
                secondResultImageGradient.y==secondResultImageGradient.y &&
                secondResultImageGradient.z==secondResultImageGradient.z)
            {
                float jointEntropyDerivative_X = 0.0f;
                float movingEntropyDerivative_X = 0.0f;
                float fixedEntropyDerivative_X = 0.0f;

                float jointEntropyDerivative_Y = 0.0f;
                float movingEntropyDerivative_Y = 0.0f;
                float fixedEntropyDerivative_Y = 0.0f;

                float jointEntropyDerivative_Z = 0.0f;
                float movingEntropyDerivative_Z = 0.0f;
                float fixedEntropyDerivative_Z = 0.0f;

                float jointLog, targetLog, resultLog, temp;
                float4 relative_pos = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                float s_x, s_y, s_z, s_w;
                float common_target_value = 0.0f;
                int target_flat_index, result_flat_index, total_target_entries, num_probabilities;
                for (int i=-1; i<2; ++i) {
                    relative_pos.x = (int)(voxelValues.x+i);

                    if (-1<relative_pos.x && relative_pos.x<c_firstTargetBin) {
                        for (int j=-1; j<2; ++j) {
                            relative_pos.y = (int)(voxelValues.y+j);

                            if (-1<relative_pos.y && relative_pos.y<c_secondTargetBin) {
                                s_x = GetBasisSplineValue(relative_pos.x-voxelValues.x);
                                s_y = GetBasisSplineValue(relative_pos.y-voxelValues.y);
                                common_target_value =  s_x * s_y;

                                for (int k=-1; k<2; ++k) {
                                    relative_pos.z = (int)(voxelValues.z+k);
                                    if (-1<relative_pos.z && relative_pos.z<c_firstResultBin) {
                                        s_x = GetBasisSplineDerivativeValue(relative_pos.z-voxelValues.z);
                                        s_w = GetBasisSplineValue(relative_pos.z-voxelValues.z);
                                        for (int l=-1; l<2; ++l) {
                                            relative_pos.w = (int)(voxelValues.w+l);
                                            if (-1<relative_pos.w && relative_pos.w<c_secondResultBin) {
                                                target_flat_index = relative_pos.x + relative_pos.y * c_firstTargetBin;
                                                result_flat_index = relative_pos.z + relative_pos.w * c_firstResultBin;
                                                total_target_entries = c_firstTargetBin * c_secondTargetBin;
                                                num_probabilities = total_target_entries * c_firstResultBin * c_secondResultBin;

                                                jointLog = tex1Dfetch(histogramTexture, target_flat_index + (result_flat_index * total_target_entries));
                                                targetLog = tex1Dfetch(histogramTexture, num_probabilities + target_flat_index);
                                                resultLog = tex1Dfetch(histogramTexture, num_probabilities + total_target_entries + result_flat_index);

                                                // Contribution from floating images. These arithmetic operations use
                                                // a lot of registers. Need to look into whether this can be reduced somehow.
                                                s_y = GetBasisSplineValue(relative_pos.w-voxelValues.w);
                                                s_z = GetBasisSplineDerivativeValue(relative_pos.w-voxelValues.w);
                                                temp = (s_x * firstResultImageGradient.x * s_y) +
                                                       (s_z * secondResultImageGradient.x * s_w);
                                                temp *= common_target_value;

                                                jointEntropyDerivative_X -= temp * jointLog;
                                                fixedEntropyDerivative_X -= temp * targetLog;
                                                movingEntropyDerivative_X -= temp * resultLog;

                                                temp = (s_x * firstResultImageGradient.y * s_y) +
                                                       (s_z * secondResultImageGradient.y * s_w);
                                                temp *= common_target_value;
                                                jointEntropyDerivative_Y -= temp * jointLog;
                                                fixedEntropyDerivative_Y -= temp * targetLog;
                                                movingEntropyDerivative_Y -= temp * resultLog;

                                                temp = (s_x * firstResultImageGradient.z * s_y) +
                                                       (s_z * secondResultImageGradient.z * s_w);
                                                temp *= common_target_value;
                                                jointEntropyDerivative_Z -= temp * jointLog;
                                                fixedEntropyDerivative_Z -= temp * targetLog;
                                                movingEntropyDerivative_Z -= temp * resultLog;
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                }

                gradValue.x = (fixedEntropyDerivative_X + movingEntropyDerivative_X - c_NMI * jointEntropyDerivative_X) / c_Entropies.z;
                gradValue.y = (fixedEntropyDerivative_Y + movingEntropyDerivative_Y - c_NMI * jointEntropyDerivative_Y) / c_Entropies.z;
                gradValue.z = (fixedEntropyDerivative_Z + movingEntropyDerivative_Z - c_NMI * jointEntropyDerivative_Z) / c_Entropies.z;
            }
        }
        voxelNMIGradientArray_d[targetIndex]=gradValue;
    }
}

__global__ void reg_smoothJointHistogramX_kernel(float *tempHistogram)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_secondTargetBin*c_firstResultBin*c_secondResultBin){
        // The starting index is computed
        unsigned int startingPoint=tid*c_firstTargetBin;
        unsigned int finishPoint=startingPoint+c_firstTargetBin;

        // The first point is computed
        tempHistogram[startingPoint] = (tex1Dfetch(histogramTexture, startingPoint) * COEFF_C +
                                       tex1Dfetch(histogramTexture, startingPoint+1) * COEFF_L) / COEFF_B;
        // The middle points are computed
        for(unsigned int i=startingPoint+1; i<finishPoint-1; ++i){
            tempHistogram[i] = tex1Dfetch(histogramTexture, i-1) * COEFF_L +
                               tex1Dfetch(histogramTexture, i) * COEFF_C +
                               tex1Dfetch(histogramTexture, i+1) * COEFF_L;
        }
        // The last point is computed
        tempHistogram[finishPoint-1] = (tex1Dfetch(histogramTexture, finishPoint-2) * COEFF_L +
                                       tex1Dfetch(histogramTexture, finishPoint-1) * COEFF_C) / COEFF_B;
    }
    return;
}

__global__ void reg_smoothJointHistogramY_kernel(float *tempHistogram)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstTargetBin*c_firstResultBin*c_secondResultBin){
        // The starting index is computed
        unsigned int startingPoint=tid + c_firstTargetBin*(c_secondTargetBin-1)*(c_firstResultBin*(int)(tid/(c_firstTargetBin*c_firstResultBin)) +
                                   (int)(tid/c_firstTargetBin - c_firstResultBin * (int)(tid/(c_firstTargetBin*c_firstResultBin))));
        unsigned int increment = c_firstTargetBin;
        unsigned int finishPoint=startingPoint+increment*c_secondTargetBin;

        // The first point is computed
        tempHistogram[startingPoint] = (tex1Dfetch(histogramTexture, startingPoint) * COEFF_C +
                                       tex1Dfetch(histogramTexture, startingPoint+increment) * COEFF_L) / COEFF_B;
        // The middle points are computed
        for(unsigned int i=startingPoint+increment; i<finishPoint-increment; i+=increment){
            tempHistogram[i] = tex1Dfetch(histogramTexture, i-increment) * COEFF_L +
                               tex1Dfetch(histogramTexture, i) * COEFF_C +
                               tex1Dfetch(histogramTexture, i+increment) * COEFF_L;
        }
        // The last point is computed
        tempHistogram[finishPoint-increment] = (tex1Dfetch(histogramTexture, finishPoint-2*increment) * COEFF_L +
                                       tex1Dfetch(histogramTexture, finishPoint-increment) * COEFF_C) / COEFF_B;
    }
    return;
}

__global__ void reg_smoothJointHistogramZ_kernel(float *tempHistogram)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstTargetBin*c_secondTargetBin*c_secondResultBin){
        // The starting index is computed
        unsigned int startingPoint=tid+c_firstTargetBin*c_secondTargetBin*(c_firstResultBin-1)*(int)(tid/(c_firstTargetBin*c_secondTargetBin));
        unsigned int increment = c_firstTargetBin*c_secondTargetBin;
        unsigned int finishPoint=startingPoint+increment*c_firstResultBin;

        // The first point is computed
        tempHistogram[startingPoint] = (tex1Dfetch(histogramTexture, startingPoint) * COEFF_C +
                                       tex1Dfetch(histogramTexture, startingPoint+increment) * COEFF_L) / COEFF_B;
        // The middle points are computed
        for(unsigned int i=startingPoint+increment; i<finishPoint-increment; i+=increment){
            tempHistogram[i] = tex1Dfetch(histogramTexture, i-increment) * COEFF_L +
                               tex1Dfetch(histogramTexture, i) * COEFF_C +
                               tex1Dfetch(histogramTexture, i+increment) * COEFF_L;
        }
        // The last point is computed
        tempHistogram[finishPoint-increment] = (tex1Dfetch(histogramTexture, finishPoint-2*increment) * COEFF_L +
                                       tex1Dfetch(histogramTexture, finishPoint-increment) * COEFF_C) / COEFF_B;
    }
    return;
}

__global__ void reg_smoothJointHistogramW_kernel(float *tempHistogram)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstTargetBin*c_secondTargetBin*c_firstResultBin){
        // The starting index is computed
        unsigned int startingPoint=tid;
        unsigned int increment = c_firstTargetBin*c_secondTargetBin*c_firstResultBin;
        unsigned int finishPoint=increment*c_secondResultBin;

        // The first point is computed
        tempHistogram[startingPoint] = (tex1Dfetch(histogramTexture, startingPoint) * COEFF_C +
                                       tex1Dfetch(histogramTexture, startingPoint+increment) * COEFF_L) / COEFF_B;
        // The middle points are computed
        for(unsigned int i=startingPoint+increment; i<finishPoint-increment; i+=increment){
            tempHistogram[i] = tex1Dfetch(histogramTexture, i-increment) * COEFF_L +
                               tex1Dfetch(histogramTexture, i) * COEFF_C +
                               tex1Dfetch(histogramTexture, i+increment) * COEFF_L;
        }
        // The last point is computed
        tempHistogram[finishPoint-increment] = (tex1Dfetch(histogramTexture, finishPoint-2*increment) * COEFF_L +
                                       tex1Dfetch(histogramTexture, finishPoint-increment) * COEFF_C) / COEFF_B;
    }
    return;
}

/// Kernels for marginalisation along the different axes
__global__ void reg_marginaliseTargetX_kernel(float *babyHisto)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_secondTargetBin*c_firstResultBin*c_secondResultBin){
        unsigned int startingPoint=tid*c_firstTargetBin;
        unsigned int finishPoint=startingPoint+c_firstTargetBin;

        float sum=tex1Dfetch(histogramTexture, startingPoint);
        float c=0.f,Y,t;
        for(unsigned int i=startingPoint+1; i<finishPoint; ++i){
            Y = tex1Dfetch(histogramTexture, i) - c;
            t = sum + Y;
            c = (t-sum)-Y;
            sum=t;
        }
        babyHisto[tid]=sum;
    }
}

__global__ void reg_marginaliseTargetXY_kernel(float *babyHisto)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstResultBin*c_secondResultBin){
        unsigned int startingPoint=tid*c_secondTargetBin;
        unsigned int finishPoint=startingPoint+c_secondTargetBin;

        float sum=tex1Dfetch(histogramTexture, startingPoint);        
        float c=0.f,Y,t;
        for(unsigned int i=startingPoint+1; i<finishPoint; ++i){            
            Y = tex1Dfetch(histogramTexture, i) - c;
            t = sum + Y;
            c = (t-sum)-Y;
            sum=t;
        }        
        babyHisto[tid]=sum;
    }
}

__global__ void reg_marginaliseResultX_kernel(float *babyHisto)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstTargetBin*c_secondTargetBin*c_firstResultBin){
        unsigned int startingPoint = tid;
        float sum=tex1Dfetch(histogramTexture, startingPoint);
        // increment by a the cube
        unsigned int increment = c_firstTargetBin*c_secondTargetBin*c_firstResultBin;
        float c=0.f,Y,t;

        for (unsigned int i = 1; i < c_secondResultBin; ++i)
        {
            Y = tex1Dfetch(histogramTexture, startingPoint + i *increment) - c;
            t = sum + Y;
            c = (t-sum)-Y;
            sum=t;
        }
        babyHisto[tid]=sum;
    }
}

__global__ void reg_marginaliseResultXY_kernel(float *babyHisto)
{
    const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(tid<c_firstTargetBin*c_secondTargetBin){
        unsigned int startingPoint=tid;
        float sum=tex1Dfetch(histogramTexture, startingPoint);
        // increment by the plane.
        unsigned int increment = c_firstTargetBin*c_secondTargetBin;
        float c=0.f,Y,t;
        for (unsigned int i = 1; i < c_firstResultBin; ++i)
        {
            Y = tex1Dfetch(histogramTexture, startingPoint + i *increment) - c;
            t = sum + Y;
            c = (t-sum)-Y;
            sum=t;
        }
        babyHisto[tid]=sum;
    }
}

#endif
